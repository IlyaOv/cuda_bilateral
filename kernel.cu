#include "hip/hip_runtime.h"
texture<unsigned int, 2, hipReadModeElementType> tex_image;

__device__ float g_func(float x, float y, float sigma)
{
	return exp(-(pow(x, 2) + pow(y, 2)) / pow(sigma, 2));
}

__device__ float r_func(float f_i, float f_0, float sigma)
{
	return exp((pow(f_i, 2) - pow(f_0, 2)) / pow(sigma, 2));
}

__global__ void bilateral_filtering_gpu(unsigned char* result, int M, int N, float sigma_d, float sigma_r)
{   
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if ((i < M) && (j < N)) {
            float h = 0;
            float k = 0;
            unsigned char f_0 = tex2D(tex_image, i, j);
            for (int j_k = i - 1; j_k <= i + 1; j_k++){
                for (int i_k = j - 1; i_k <= j + 1; i_k++){
                    unsigned char f_i = tex2D(tex_image, i_k, j_k);
                    float g = g_func((i_k - i), (j_k - j), sigma_d);
                    float r = r_func(f_i, f_0, sigma_r);
                    k += g * r;
                    h += g * r * tex2D(tex_image, i_k, j_k);
                }
            }
            result[i * N + j] = h / k;
        }
}